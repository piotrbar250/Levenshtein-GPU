#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string.h>
#include <stack>

#include "Utils.h"
#include "Timer.h"
#include "Solutions.h"

using namespace std;

int X[AL][M];

__host__ __device__ int min(int a, int b, int c)
{
	if (a <= b && a <= c)
		return a;
	if (b <= a && b <= c)
		return b;
	return c;
}

__global__ void computeXKernel(int X[AL][M], char* text, int m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < AL) {
		X[i][0] = 0;
		for (int j = 1; j <= m; j++)
		{
			if (i + 65 == (int)text[j - 1])
				X[i][j] = j;
			else
				X[i][j] = X[i][j - 1];
		}
	}
}

__global__ void computeDpKernel(int dp[N][M], int X[AL][M], char* pattern, char* text, int n, int m, int i)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i <= n && j <= m)
	{
		if (i == 0)
			dp[i][j] = j;
		else if (j == 0)
			dp[i][j] = i;
		else if (text[j - 1] == pattern[i - 1])
			dp[i][j] = dp[i - 1][j - 1];
		else
		{
			int l = ((int)pattern[i - 1] - 65);
			if (X[l][j] == 0)
			{
				dp[i][j] = 1 + min(dp[i - 1][j], dp[i - 1][j - 1], i + j - 1);
			}
			else
			{
				dp[i][j] = 1 + min(dp[i - 1][j], dp[i - 1][j - 1], dp[i - 1][X[l][j] - 1] + (j - 1 - X[l][j]));
			}
		}
	}
}

void solveGPU(string p, string t, int dp[N][M])
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	int n = p.size();
	int m = t.size();

	int(*device_X)[M];
	cudaStatus = hipMalloc((void**)&device_X, AL * M * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	char* textC = const_cast<char*>(t.c_str());
	char* device_text;
	cudaStatus = hipMalloc((void**)&device_text, m + 1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(device_text, textC, m + 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	char* patternC = const_cast<char*>(p.c_str());
	char* device_pattern;
	cudaStatus = hipMalloc((void**)&device_pattern, n + 1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_pattern, patternC, n + 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	computeXKernel << <1, AL >> > (device_X, device_text, m);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeXKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipDeviceSynchronize();
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeXKernel!\n", cudaStatus);
		goto Error;
	}

	int(*d_dp)[M];
	cudaStatus = hipMalloc((void**)&d_dp, N * M * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 threadsPerBlock(256);
	dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

	for (int i = 0; i <= n; i++)
	{
		computeDpKernel << <numBlocks, threadsPerBlock >> > (d_dp, device_X, device_pattern, device_text, n, m, i);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "computeDpKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		hipDeviceSynchronize();
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeDpKernel!\n", cudaStatus);
			goto Error;
		}
	}

	cudaStatus = hipMemcpy(dp, d_dp, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	Error:
	hipFree(device_X);
	hipFree(device_pattern);
	hipFree(d_dp);
}