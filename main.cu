#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string.h>

using namespace std;

#define N 1000
#define M 1000
#define AL 26
int dp[N][M];
int X[AL][M];

int n, m;

void print()
{
	for (int i = 0; i <= n; i++)
	{
		for (int j = 0; j <= m; j++)
			cout << dp[i][j] << " ";
		cout << endl;
	}
}

void printX()
{
	for (int i = 0; i < AL; i++)
	{
		cout << (char)(i + 65) << " ";
		for (int j = 0; j <= m; j++)
			cout << X[i][j] << " ";
		cout << endl;
	}
}

__device__ int min(int a, int b, int c)
{
	if (a <= b && a <= c)
		return a;
	if (b <= a && b <= c)
		return b;
	return c;
}

__global__ void testKernel(int dp[N][M], int n, int m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i <= n && j <= m) {
		dp[i][j] = i * (m + 1) + j;
	}
}

__global__ void computeXKernel(int X[AL][M], char* text, int m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// i powinno byc od 0 do 25z
	if (i < AL) {
		X[i][0] = 0;
		for (int j = 1; j <= m; j++)
		{
			if (i + 65 == (int)text[j - 1])
				X[i][j] = j;
			else
				X[i][j] = X[i][j - 1];
		}
	}
}

__global__ void computeDpKernel(int dp[N][M], int X[AL][M], char* pattern, char* text, int n, int m, int i)
{
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i <= n && j <= m) {
		if (i == 0)
			dp[i][j] = 0;
		else if (j == 0)
			dp[i][j] = i;
		else if (text[j - 1] == pattern[i - 1])
			dp[i][j] = dp[i - 1][j - 1];
		else
		{
			int l = ((int)pattern[i - 1] - 65);
			if (X[l][j] == 0)
				dp[i][j] = 1 + min(dp[i - 1][j], dp[i - 1][j - 1], i + j - 1);
			else
				dp[i][j] = 1 + min(dp[i - 1][j], dp[i - 1][j - 1], dp[i - 1][X[l][j] - 1 + (j - 1 - X[l][j])]);
		}
	}
}

int main()
{
	string t = "tekstowo";
	string p = "pattern";

	t = "1234567";
	p = "123";

	t = "CATGACTG";
	p = "TACTG";

	n = p.size();
	m = t.size();

	int(*device_X)[M];
	hipMalloc((void**)&device_X, AL * M * sizeof(int));

	char* textC = const_cast<char*>(t.c_str());
	char* device_text;
	hipMalloc((void**)&device_text, m + 1);
	hipMemcpy(device_text, textC, m + 1, hipMemcpyHostToDevice);

	char* patternC = const_cast<char*>(p.c_str());
	char* device_pattern;
	hipMalloc((void**)&device_pattern, n + 1);
	hipMemcpy(device_pattern, patternC, n + 1, hipMemcpyHostToDevice);

	computeXKernel << <1, AL >> > (device_X, device_text, m);
	hipDeviceSynchronize();

	hipMemcpy(X, device_X, sizeof(int) * AL * M, hipMemcpyDeviceToHost);

	printX();

	// ---------------------------------------------------------------------------------------------------------------

	int(*d_dp)[M];

	hipMalloc((void**)&d_dp, N * M * sizeof(int));
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(M + threadsPerBlock.y - 1) / threadsPerBlock.y);

	testKernel << <numBlocks, threadsPerBlock >> > (d_dp, n, m);

	for (int i = 0; i <= n; i++)
	{
		computeDpKernel << <numBlocks, threadsPerBlock >> > (d_dp, device_X, device_pattern, device_text, n, m, i);
		hipDeviceSynchronize();
	}

	hipMemcpy(dp, d_dp, sizeof(int) * N * M, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	print();

}